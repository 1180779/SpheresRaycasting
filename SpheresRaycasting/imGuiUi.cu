#include "hip/hip_runtime.h"

#include "imGuiUi.cuh"

imGuiUi::imGuiUi(rendering& rendering) : m_rendering(rendering), io((ImGui::CreateContext(), ImGui::GetIO()))
{
    // Setup Dear ImGui context
    IMGUI_CHECKVERSION();
    io.ConfigFlags |= ImGuiConfigFlags_NavEnableKeyboard;     // Enable Keyboard Controls
    io.ConfigFlags |= ImGuiConfigFlags_NavEnableGamepad;      // Enable Gamepad Controls

    // Setup Platform/Renderer backends
    ImGui_ImplGlfw_InitForOpenGL(m_rendering.window, true);
#ifdef __EMSCRIPTEN__
    ImGui_ImplGlfw_InstallEmscriptenCallbacks(window, "#canvas");
#endif
    ImGui_ImplOpenGL3_Init(m_rendering.glsl_version);
}

imGuiUi::~imGuiUi()
{
    ImGui_ImplOpenGL3_Shutdown();
    ImGui_ImplGlfw_Shutdown();
    ImGui::DestroyContext();
}

void imGuiUi::styleRounded()
{
    ImGuiStyle& style = ImGui::GetStyle();
    style.TabRounding = 8.f;
    style.FrameRounding = 8.f;
    style.GrabRounding = 8.f;
    style.WindowRounding = 8.f;
    style.PopupRounding = 8.f;
}

void imGuiUi::styleSquare()
{
    ImGuiStyle& style = ImGui::GetStyle();
    style.TabRounding = 0.f;
    style.FrameRounding = 0.f;
    style.GrabRounding = 0.f;
    style.WindowRounding = 0.f;
    style.PopupRounding = 0.f;
}

void imGuiUi::styleLight()
{
    ImGui::StyleColorsLight();
}

void imGuiUi::styleDark()
{
    ImGui::StyleColorsDark();
}

static void rangeControl(const char* nameMin, const char* nameMax, range& r)
{
    ImGui::SetNextItemWidth(150);
    if (ImGui::InputFloat(nameMin, &r.min, 10.0f, 100.0f))
    {
        if (r.min > r.max)
            r.min = r.max;
    }
    ImGui::SetNextItemWidth(150);
    if (ImGui::InputFloat(nameMax, &r.max, 10.0f, 100.0f))
    {
        if (r.max < r.min)
            r.max = r.min;
    }
}

static void rangeControlWithLimits(const char* nameMin, const char* nameMax, range& r, range limits)
{
    ImGui::SetNextItemWidth(150);
    if (ImGui::InputFloat(nameMin, &r.min, 0.05f, 0.2f))
    {
        if (r.min > r.max)
            r.min = r.max;
        if (r.min < limits.min)
            r.min = limits.min;
    }
    ImGui::SetNextItemWidth(150);
    if (ImGui::InputFloat(nameMax, &r.max, 0.05f, 0.2f))
    {
        if (r.max < r.min)
            r.max = r.min;
        if (r.max > limits.max)
            r.max = limits.max;
    }
}

static void countControl(const char* name, unsigned int& c) 
{
    ImGui::SetNextItemWidth(150);
    if (ImGui::InputInt(name, (int*)&c, 1, 10))
    {
        if ((int)c < 0)
            c = 0;
    }
}
void imGuiUi::constSettingsWindow(bool& start, sceneConfig& config, int& camWidth, int& camHeight)
{
    /* start window */
    constexpr float spacing = 15.0f;
    ImGuiWindowFlags winFlags = ImGuiWindowFlags_NoResize | ImGuiWindowFlags_NoMove | ImGuiWindowFlags_NoCollapse;

    /* sphere settings window */
    ImGui::SetNextWindowPos(ImVec2(20, 20));
    ImGui::SetNextWindowSize(ImVec2(400, 620));
    ImGui::Begin("Sphere settings", NULL, winFlags);
    
    ImGui::Dummy(ImVec2(0.0f, spacing));
    countControl("spheres count", config.sCount);
    rangeControl("spheres x range min", "spheres x range max", config.sXR);
    rangeControl("spheres y range min", "spheres y range max", config.sYR);
    rangeControl("spheres z range min", "spheres z range max", config.sZR);
    rangeControl("spheres radius range min", "spheres radius range max", config.sRR);

    ImGui::Dummy(ImVec2(0.0f, spacing));
    ImGui::Combo("material type", (int*)(&config.matType), 
        materialGenerator::typeString, materialGenerator::typeCount);
    ImGui::End();

    /* light settings window */
    ImGui::SetNextWindowPos(ImVec2(440, 20));
    ImGui::SetNextWindowSize(ImVec2(400, 620));
    ImGui::Begin("Light settings", NULL, winFlags);
    ImGui::Dummy(ImVec2(0.0f, spacing));
    countControl("lights count", config.lCount);
    rangeControl("lights x range min", "lights x range max", config.lXR);
    rangeControl("lights y range min", "lights y range max", config.lYR);
    rangeControl("lights z range min", "lights z range max", config.lZR);
    rangeControl("lights radius range min", "lights radius range max", config.lRR);

    rangeControlWithLimits("light Is min", "light Is max", config.isR, range(0.0f, 1.0f));
    rangeControlWithLimits("light Id min", "light Id max", config.idR, range(0.0f, 1.0f));
    ImGui::End();


    /* other settings and start window */
    ImGui::SetNextWindowPos(ImVec2(860, 20));
    ImGui::SetNextWindowSize(ImVec2(400, 200));
    ImGui::Begin("Settings", NULL, winFlags);
    ImGui::SetNextItemWidth(150);
    ImGui::ColorEdit3("Change background", (float*)&m_rendering.clear_color);

    ImGui::Text("Camera resolution");
    ImGui::SetNextItemWidth(150);
    if (ImGui::InputInt("camera res x", (int*)&camWidth, 10, 100))
    {
        if (camWidth > 1920)
            camWidth = 1920;
        if (camWidth < 1280)
            camWidth = 1280;
    }
    ImGui::SetNextItemWidth(150);
    if (ImGui::InputInt("camera res y", (int*)&camHeight, 10, 100))
    {
        if (camHeight > 1080)
            camHeight = 1080;
        if (camHeight < 720)
            camHeight = 720;
    }

    ImGui::Dummy(ImVec2(0.0f, spacing));
    start = ImGui::Button("Start");
    if(ImGui::Button("Load from file")) 
    {
        config.loadFromFile("config.txt");
    }

    ImGui::Text("Average %.3f ms/frame (%.1f FPS)", 1000.0f / io.Framerate, io.Framerate);
    ImGui::End();
}

void imGuiUi::settingsWindow(float& ia, bool& animate)
{
    ImGui::SetNextWindowSize(ImVec2(300, 200));
    ImGui::Begin("Dynamic settings", NULL, ImGuiWindowFlags_NoResize);

    ImGui::SetNextItemWidth(150);
    ImGui::ColorEdit3("Change background", (float*)&m_rendering.clear_color);

    ImGui::Checkbox("Animate", &animate);

    ImGui::BeginGroup();
    ImGui::Text("Rotation mode");
    if(ImGui::RadioButton("spheres", m_rotateObjects))
    {
        m_rotateObjects = true;
    }

    if(ImGui::RadioButton("lights", !m_rotateObjects))
    {
        m_rotateObjects = false;
    }
    ImGui::EndGroup();

    ImGui::SetNextItemWidth(150);
    if(ImGui::InputFloat("Ia (ambient)", &ia, 0.005f, 0.01f)) 
    {
        if (ia < 0.0f)
            ia = 0.0f;
        if (ia > 1.0f)
            ia = 1.0f;
    }

    ImGui::Text("Average %.3f ms/frame (%.1f FPS)", 1000.0f / io.Framerate, io.Framerate);
    ImGui::Text("Press ESC to exit to ui");
    ImGui::End();
}

void imGuiUi::checkInput()
{
    m_inputEscape = false;
    m_inputMouseClicked = false;
    m_inputMouseInView = true;
    if(ImGui::IsKeyPressed(ImGuiKey_Escape))
    {
        m_inputEscape = true;
    }
    if(ImGui::IsMouseClicked(ImGuiMouseButton_Left)) 
    {
        m_inputMouseClicked = true;
    }
}

void imGuiUi::handleInput()
{
    if(m_inputMouseLocked) 
    {
        if(m_inputEscape) 
        {
            /* release the mouse, reenable glfw mouse callback */
            glfwSetInputMode(m_rendering.window, GLFW_CURSOR, GLFW_CURSOR_NORMAL);
            glfwSetCursorPosCallback(m_rendering.window, ImGui_ImplGlfw_CursorPosCallback);
            m_inputMouseLocked = false;
        }
    }
    else 
    {
        if(m_inputMouseClicked && !io.WantCaptureMouse) 
        {
            /* capture the mouse */
            glfwSetInputMode(m_rendering.window, GLFW_CURSOR, GLFW_CURSOR_DISABLED);
            m_inputMouseLocked = true;
            if(m_rotateObjects) 
            {
                
                glfwSetCursorPosCallback(m_rendering.window, mouseCallbackRotateAll);
            }
            else 
            {
                glfwSetCursorPosCallback(m_rendering.window, mouseCallbackRotateLights);
            }
        }
    }
}

void imGuiUi::newFrame()
{
    ImGui_ImplOpenGL3_NewFrame();
    ImGui_ImplGlfw_NewFrame();
    ImGui::NewFrame();
}
