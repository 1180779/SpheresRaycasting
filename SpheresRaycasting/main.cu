﻿
/* CUDA */
#include <hip/hip_runtime.h>
#include <>

#include "castRays.cuh"

/* NOT CUDA */
#include "rendering.hpp"
#include "imGuiUi.hpp"

#include "shader.hpp"
#include "shadersC.hpp"
#include "camera.hpp"

#include "camera.hpp"
#include "objectList.hpp"


#include "triangleShape.hpp"
#include "rectangleShape.hpp"
#include "cubeShape.hpp"



#define STB_IMAGE_IMPLEMENTATION

int main(int, char**)
{
    rendering render = rendering();
    imGuiUi ui = imGuiUi(render);
    ui.styleLight();
    ui.styleRounded();
    render.initGL();
    glEnable(GL_DEPTH_TEST);


    camera cam(render);
    cam.setCurrent();
    //camera::disableCursor(render);
    //camera::setCallbacks(render);


    // Main loop
    while (!glfwWindowShouldClose(render.window))
    {
        if (glfwGetKey(render.window, GLFW_KEY_ESCAPE)) {
            glfwSetWindowShouldClose(render.window, GL_TRUE);
        }
        render.measureDeltaTime();
        cam.processInput();
        // Poll and handle events (inputs, window resize, etc.)
        // You can read the io.WantCaptureMouse, io.WantCaptureKeyboard flags to tell if dear imgui wants to use your inputs.
        // - When io.WantCaptureMouse is true, do not dispatch mouse input data to your main application, or clear/overwrite your copy of the mouse data.
        // - When io.WantCaptureKeyboard is true, do not dispatch keyboard input data to your main application, or clear/overwrite your copy of the keyboard data.
        // Generally you may always pass all inputs to dear imgui, and hide them from your application based on those two flags.
        glfwPollEvents();
        if (glfwGetWindowAttrib(render.window, GLFW_ICONIFIED) != 0)
        {
            ImGui_ImplGlfw_Sleep(10);
            continue;
        }

        ui.newFrame();
        ui.settingsWindow();

        // Rendering
        ImGui::Render();
        int display_w, display_h;
        glfwGetFramebufferSize(render.window, &display_w, &display_h);
        glViewport(0, 0, display_w, display_h);
        render.clearColor();
        glClear(GL_DEPTH_BUFFER_BIT);


        ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());
        render.swapBuffers();
    }
    return 0;
}