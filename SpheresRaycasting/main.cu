﻿
/* CUDA */
#include <hip/hip_runtime.h>
#include <>

#include "castRays.cuh"
#include "buffer.cuh"
#include "general.hpp"

#include "callbacks.cuh"

/* NOT CUDA */
#include "rendering.hpp"
#include "imGuiUi.hpp"

#include "shader.hpp"

#include "mat4.cuh"

#include "dataObject.hpp"

void matTests() {
    glm::mat4 tGLM = glm::rotate(glm::mat4(1.0f), 180.0f, glm::vec3(1.0f, 0.0f, 0.0f));
    mat4 t;
    t = tGLM;

    glm::vec4 vGLM(1.f, 1.f, 1.f, 1.f);
    vec4 v(1.f, 1.f, 1.f, 1.f);

    v = t * v;
    vGLM = tGLM * vGLM;

    std::cout << "x = " << v(0) << ", y = " << v(1) << ", z = " << v(2) << ", wGLM = " << v(3) << std::endl;
    std::cout << "xGLM = " << vGLM[0] << ", yGLM = " << vGLM[1] << ", zGLM = " << vGLM[2] << ", wGLM = " << vGLM[3] << std::endl;
}

int main(int, char**)
{
    //matTests();
    //return;

    rendering render = rendering();
    imGuiUi ui = imGuiUi(render);
    ui.styleLight();
    ui.styleRounded();
    render.initGL();

    xcudaSetDevice(0);

    disableCursor(render);

    buffer b = buffer();


    dataObject data;
    data.generate(200, 50, 50, -1920, 1920, -1080, 1080, 100, 200);

    castRaysData raysData;
    raysData.data = data.md_unified;
   
    transformData tData;
    tData.data = data.md_unified;

    spheresDataForCallback = &tData;

    dim3 blocksForSpheres = dim3(data.dCount() / BLOCK_SIZE + 1);
    dim3 threadsForSpheres = dim3(BLOCK_SIZE);

    dim3 blocks = dim3(b.m_maxWidth / BLOCK_SIZE + 1, b.m_maxHeight / BLOCK_SIZE + 1);
    dim3 threads = dim3(BLOCK_SIZE, BLOCK_SIZE);


    // Main loop
    while (!glfwWindowShouldClose(render.window))
    {
        if (glfwGetKey(render.window, GLFW_KEY_ESCAPE)) {
            glfwSetWindowShouldClose(render.window, GL_TRUE);
        }
        render.measureDeltaTime();
        // Poll and handle events (inputs, window resize, etc.)
        // You can read the io.WantCaptureMouse, io.WantCaptureKeyboard flags to tell if dear imgui wants to use your inputs.
        // - When io.WantCaptureMouse is true, do not dispatch mouse input data to your main application, or clear/overwrite your copy of the mouse data.
        // - When io.WantCaptureKeyboard is true, do not dispatch keyboard input data to your main application, or clear/overwrite your copy of the keyboard data.
        // Generally you may always pass all inputs to dear imgui, and hide them from your application based on those two flags.
        glfwPollEvents();
        if (glfwGetWindowAttrib(render.window, GLFW_ICONIFIED) != 0)
        {
            ImGui_ImplGlfw_Sleep(10);
            continue;
        }

        ui.newFrame();
        ui.settingsWindow();

        // Rendering
        ImGui::Render();
        int display_w, display_h;
        glfwGetFramebufferSize(render.window, &display_w, &display_h);
        glViewport(0, 0, display_w, display_h);
        render.clearColor();
        glClear(GL_DEPTH_BUFFER_BIT);


        b.mapCudaResource();


        raysData.width = b.m_maxWidth;
        raysData.height = b.m_maxHeight;
        raysData.surfaceObject = b.m_surfaceObject;
        castRaysKernel << <blocks, threads >> > (raysData);
        xcudaDeviceSynchronize();
        xcudaGetLastError();

        //data.mh_spheres.copyDeviceToHost(raysData.sData);
        //std::cout << "\n\n";
        //std::cout << "DATA" << std::endl;
        //for (int i = 0; i < data.mh_spheres.count; ++i) {
        //    std::cout << "x = " << data.mh_spheres.x[i] << ", y = " << data.mh_spheres.y[i] << ", z = " << data.mh_spheres.z[i] << ", r = " << data.mh_spheres.r[i] << std::endl;
        //}

        b.unmapCudaResource();
        b.use();

        ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());
        render.swapBuffers();
    }
    data.free();
    return 0;
}